#include "hip/hip_runtime.h"
#include <iostream>
#include "lodepng.h"

/* Kernel for inverting usigned char */
__global__ void invert_image( unsigned char *image ) {
    int index = threadIdx.x + blockIdx.x * blockDim.x; 
    image[index] = ~image[index]; /* Invert unsigned char */
}

# define THREADS_PER_BLOCK 512

int main( int argc, char ** argv){

    size_t pngsize;
    unsigned char *png;
    const char * filename = "lenna512x512_inv.png";
    /* Read in the image */
    lodepng_load_file(&png, &pngsize, filename);

    unsigned char *image; /* Host copy of image */
    unsigned int width, height;
    /* Decode it into a RGB 8-bit per channel vector */
    unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

    /* Check if read and decode of .png went well */
    if(error != 0){
        std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
    }
	
    // Do work
	unsigned char *device_image;
	int loop_size = width * height * 3;
	size_t size = loop_size * sizeof( unsigned char );

	hipMalloc( ( void** ) &device_image, size );
	hipMemcpy( device_image, image, size, hipMemcpyHostToDevice );

    invert_image<<<loop_size / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>( device_image );
	
    hipMemcpy( image, device_image, size, hipMemcpyDeviceToHost );

    /* Save the result to a new .png file */
    lodepng_encode24_file("lenna512x512_orig.png", image , width,height);

	free( image );
	hipFree( device_image );

    return 0;
}

